#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "StructsHeader.h"


__global__ void movePoints(Point* devPoints, int allPointsSize, unsigned int numofThreadsperBlock, unsigned int numofBlocks, double theTime)
{
	unsigned int index = blockIdx.x * numofThreadsperBlock + threadIdx.x;
	unsigned int threadwork = allPointsSize / ((numofThreadsperBlock)*(numofBlocks));

	for (unsigned int i = index*threadwork; i < (index* threadwork) + threadwork; i++)
	{
		
			devPoints[i].x = devPoints[i].x + theTime*devPoints[i].Vx;
			devPoints[i].y = devPoints[i].y + theTime*devPoints[i].Vy;
			devPoints[i].z = devPoints[i].z + theTime*devPoints[i].Vz;
		
	}
}

//This function is the Bridge between Host Code & GPU.

hipError_t pointsLocationWholeCuda(int allPointsSize, double theTime, Point* pointsArray, unsigned int threadsperblock, int* remainderOffset)
{

	Point* devPoints = NULL;
	unsigned int numofBlocks = allPointsSize / threadsperblock;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(devPoints);
	}
	// Allocated devPoints inside GPU
	cudaStatus = hipMalloc((void**)&devPoints, allPointsSize * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(devPoints);

	}
	// Copy input vector (pointsArray) from host memory (CPU) to GPU's buffer devPoints.
	cudaStatus = hipMemcpy(devPoints, pointsArray, allPointsSize * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(devPoints);
	}

	// Launch a kernel on the GPU
	movePoints << <numofBlocks, threadsperblock >> > (devPoints, allPointsSize, threadsperblock, numofBlocks, theTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsLocation launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(devPoints);

	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(devPoints);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pointsArray, devPoints, allPointsSize * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(devPoints);
	}

 
	hipFree(devPoints);
	*remainderOffset = numofBlocks*threadsperblock;
	return cudaStatus;
}


